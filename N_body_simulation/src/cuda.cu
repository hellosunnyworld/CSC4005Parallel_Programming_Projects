#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/logger.h"


int block_size = 512;


int n_body;
int n_iteration;


__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {
    //TODO: update position 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        x[i] += vx[i] * dt;
        y[i] += vy[i] * dt;

        if (x[i] > 2000.0f + bound_x / 4)
        x[i] = 2000.0f + bound_x / 4 - 1;
        if (y[i] > 2000.0f + bound_y / 4)
        y[i] = 2000.0f + bound_y / 4 - 1;          
        if (x[i] < 2000.0f - bound_x / 4)
        x[i] = 2000.0f - bound_x / 4 + 1;
        if (y[i] < 2000.0f - bound_y / 4)
        y[i] = 2000.0f - bound_y / 4 + 1;  
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    //TODO: calculate force and acceleration, update velocity
    int pi = blockDim.x * blockIdx.x + threadIdx.x;
    if (pi < n) {  
        double ori_vx;
        double ori_vy;
        ori_vx = vx[pi];
        ori_vy = vy[pi];
        for (int pj = 0; pj < n; pj++){
            if (pi != pj){
                //double r_2 = std::norm((x[pi] - x[pj], y[pi] - y[pj]));
                double r_2 = (pow(x[pi] - x[pj],2)+pow(y[pi] - y[pj],2));
                double ai = gravity_const * m[pj] / (r_2 + err);
                double ai_x = ai / (sqrt(r_2) + err) * (- x[pi] + x[pj]);
                double ai_y = ai / (sqrt(r_2) + err) * (- y[pi] + y[pj]);
                // std::cout << ((pow(ai_x,2) + pow(ai_y,2))) << std::endl;
                // std::cout << (pow(ai,2)) << std::endl;
                // std::cout << "ai_x " << ai_x << std::endl;
                // std::cout << "ai_y " << ai_y << std::endl;

                
                vx[pi] += ai_x * dt;
                vy[pi] += ai_y * dt;
                // std::cout << "vi_x " << vx[pi] << std::endl;
                // std::cout << "vi_y " << vy[pi] << std::endl;
                // std::cout << "-----------------------" << std::endl;
                if (sqrt(r_2) <= 2 * sqrt(radius2)){// collision
                    double vec_r_xi = -x[pi] + x[pj];
                    double vec_r_yi = -y[pi] + y[pj];

                    double vec_v_xi = x[pi] + vx[pi];
                    double vec_v_yi = y[pi] + vy[pi];
                    double ci_2 = pow(vec_v_xi-x[pj],2)+pow(vec_v_yi-y[pj],2);
                    double vi = sqrt(pow(vx[pi],2)+pow(vy[pi],2));
                    double cos_ai =  (- ci_2 + r_2 + pow(vi,2)) / (2 * sqrt(r_2) * vi + err);
                    double vi_changed = vi * cos_ai;
                    //double vi_changed_ = vi * sqrt(1-pow(cos_ai,2));

                    double vi_changed_x = - vi_changed * vec_r_xi / (vi + err);
                    double vi_changed_y = - vi_changed * vec_r_yi / (vi + err);
                    double vi_changed_x_ = vx[pi] + vi_changed_x;
                    double vi_changed_y_ = vy[pi] + vi_changed_y;

                    vx[pi] = vi_changed_x_ + vi_changed_x;
                    vy[pi] = vi_changed_y_ + vi_changed_y;
                }

            }
        }
        // wall collision
        if (x[pi] >= 2000.0f + bound_x / 4 - 1){
            vx[pi] -= 2 * ori_vx;
        }
        if (y[pi] >= 2000.0f + bound_y / 4 - 1){
            vy[pi] -= 2 * ori_vy;
        }
        if (x[pi] <= 2000.0f - bound_x / 4 + 1){
            vx[pi] -= 2 * ori_vx;
        }
        if (y[pi] <= 2000.0f - bound_y / 4 + 1){
            vy[pi] -= 2 * ori_vy;
        }
    }
}


void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // TODO: Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        x[i] = 2000.0f + rand() % (bound_x / 4);
        y[i] = 2000.0f + rand() % (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}



void master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);

    //Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;
    double t_span = 0;
    for (int i = 0; i < n_iteration; i++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);

        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        
        //printf("Iteration %d, elapsed time: %.3f\n", i, time_span);
        t_span += time_span.count();
        //l.save_frame(x, y);

        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete[] m;
    delete[] x;
    delete[] y;
    delete[] vx;
    delete[] vy;
    // printf("%f ",t_span/n_iteration);
    // if (n_body == 10000)
    //     printf("\\\\ \n");
    // else
    //     printf("& ");
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif

    master();

    printf("Student ID: 119010177\n"); // replace it with your student id
    printf("Name: Muhan Lin\n"); // replace it with your name
    printf("Assignment 2: N Body Simulation CUDA Implementation\n");

    return 0;

}


